#include "hip/hip_runtime.h"
// This is mostly a test
#include <GL/glew.h>
#include <GL/glut.h>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <immintrin.h>
#include <iostream>
#include <memory>
#include <stdio.h>
#include <thread>

int times = 0;
int xres = 1920, yres = 1080;
GLuint pbo = 0; // pixelbuffer obj
GLuint tex = 0; // texture obj
GLuint fbo = 0; // framebuffer obj
GLuint rbo = 0; // framebuffer obj
struct hipGraphicsResource* cuda_pbo_resource;
struct uchar4;

__global__ void calc(uchar4* rgba, int maxiter, float recdiv, float imcdiv, float cx0, float cy0,
    int xres, float* hist, int* iterimg, int& total);
__global__ void color(uchar4* rgba, float* hist, int xres, int* iterimg, int& total);
void pgm(int maxiter, int* img, int xres, int& yres);
void startkernel(uchar4* rgba);
void pboinit();

void display() {
  // create uchar and do math
  uchar4* rgba = nullptr;
  hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
  hipGraphicsResourceGetMappedPointer((void**)&rgba, NULL, cuda_pbo_resource);
  startkernel(rgba);
  hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);

  glBindTexture(GL_TEXTURE_2D, tex);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
  glBufferData(GL_PIXEL_UNPACK_BUFFER, xres * yres * 4 * sizeof(GLubyte), 0, GL_STREAM_DRAW);

  glTexSubImage2D(
      GL_TEXTURE_2D, 0, 0, 0, xres, yres, GL_RGBA, GL_UNSIGNED_BYTE, 0); // Null or pbo? hm
  glEnable(GL_TEXTURE_2D);
  glBegin(GL_QUADS);
  glTexCoord2f(0.0f, 0.0f);
  glVertex2f(0, 0);
  glTexCoord2f(0.0f, 1.0f);
  glVertex2f(0, yres);
  glTexCoord2f(1.0f, 1.0f);
  glVertex2f(xres, yres);
  glTexCoord2f(1.0f, 0.0f);
  glVertex2f(xres, 0);
  glEnd();
  glDisable(GL_TEXTURE_2D);

  glutSwapBuffers();
}

void initglut(int* argc, char** argv) {
  glutInit(argc, argv);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
  glutInitWindowSize(xres, yres);
  glutCreateWindow("mandelbrot");
  glewInit();
  pboinit();
}

void pboinit() {
  glGenBuffers(1, &pbo);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
  glBufferData(GL_PIXEL_UNPACK_BUFFER, xres * yres * 4 * sizeof(GLubyte), 0, GL_STREAM_DRAW);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
  hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);

  glGenTextures(1, &tex);
  glBindTexture(GL_TEXTURE_2D, tex);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, xres, yres, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
  glBindTexture(GL_TEXTURE_2D, 0);
}

void exitfunc() {
  if (pbo) {
    hipGraphicsUnregisterResource(cuda_pbo_resource);
    glDeleteBuffers(1, &pbo);
    glDeleteTextures(1, &tex);
  }
}

void startkernel(uchar4* rgba) {
  int maxiter = 4096;
  float const cx0 = -2, cx1 = 1, cy0 = -1.2, cy1 = 1.2;
  float const cw = cx1 - cx0, ch = cy1 - cy0;
  float const recdiv = cw / float(xres), imcdiv = ch / float(yres);
  dim3 const dimBlock(32, 8);
  dim3 const dimGrid(std::ceil(float(xres) / dimBlock.x), std::ceil(float(yres) / dimBlock.y));

  float* hist;
  int* img;
  int* total;
  hipMalloc(&hist, xres * yres * sizeof(float));
  hipMalloc(&img, xres * yres * sizeof(int));
  hipMalloc(&total, sizeof(int));

  calc<<<dimGrid, dimBlock>>>(rgba, maxiter, recdiv, imcdiv, cx0, cy0, xres, hist, img, *total);
  color<<<dimGrid, dimBlock>>>(rgba, hist, xres, img, *total);

  hipFree(hist);
  hipFree(img);
  hipFree(total);
}

__global__ void calc(uchar4* rgba, int maxiter, float recdiv, float imcdiv, float cx0, float cy0,
    int xres, float* hist, int* img, int& total) {
  int iters = 0;
  float zmag2 = 0, zrsq = 0, zisq = 0, zreal = 0, zimag = 0;
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int ind = y * xres + x;
  float creal = x * recdiv + cx0;
  float cimag = y * imcdiv + cy0;

  while (iters <= maxiter && zmag2 <= 4) {
    zimag = ((zreal + zreal) * zimag) + cimag;
    zreal = (zrsq - zisq) + creal;
    zisq = zimag * zimag;
    zrsq = zreal * zreal;
    zmag2 = zrsq + zisq;
    iters++;
  }

  hist[iters]++;
  img[ind] = iters;
}

__global__ void color(uchar4* rgba, float* hist, int xres, int* img, int& total) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int ind = y * xres + x;
  float hue;
  total = 0;

  // TODO figure out why total changes
  float res = 96000;
  for (int i = 0; i <= img[ind]; i++) {
    hue += hist[i] / res;
  }

  rgba[ind].y = hue * 255;
  rgba[ind].x = 0;   // R
  rgba[ind].z = 0;   // B
  rgba[ind].w = 255; // A
}

void mandelbrot(int argc, char** argv) {
  // GLUT STUFF
  initglut(&argc, argv);
  gluOrtho2D(0, xres, yres, 0);
  glutDisplayFunc(display);
  glutMainLoop();
  atexit(exitfunc);
}

int main(int argc, char** argv) { mandelbrot(argc, argv); }
