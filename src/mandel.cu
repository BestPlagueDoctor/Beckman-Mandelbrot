#include "hip/hip_runtime.h"
// This is mostly a test
// take fasdf asdfas lmao
#define GL_GLEXT_PROTOTYPES
#include <GL/glew.h>
#include <GL/glut.h>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <immintrin.h>
#include <iostream>
#include <memory>
#include <new>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <thread>

int xres = 1920, yres = 1080;
GLuint pbo = 0; // pixelbuffer obj
GLuint tex = 0; // texture obj
struct hipGraphicsResource* cuda_pbo_resource;
struct uchar4;

__global__ void calc(
    uchar4* dout, int maxiter, float recdiv, float imcdiv, float cx0, float cy0, int xres);
void pgm(int maxiter, int* img, int xres, int yres);
void startkernel(uchar4* dout);

void render() {
  uchar4* dout = 0;
  hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
  hipGraphicsResourceGetMappedPointer((void**)&dout, NULL, cuda_pbo_resource);
  startkernel(dout);
  hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
}

void drawTex() {
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, xres, yres, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
  glEnable(GL_TEXTURE_2D);
  glBegin(GL_QUADS);
  glTexCoord2f(0.0f, 0.0f);
  glVertex2f(0, 0);
  glTexCoord2f(0.0f, 1.0f);
  glVertex2f(0, yres);
  glTexCoord2f(1.0f, 1.0f);
  glVertex2f(xres, yres);
  glTexCoord2f(1.0f, 0.0f);
  glVertex2f(xres, 0);
  glEnd();
  glDisable(GL_TEXTURE_2D);
}

void display() {
  render();
  drawTex();
  glutSwapBuffers();
}

void initglut(int* argc, char** argv) {
  glutInit(argc, argv);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
  glutInitWindowSize(xres, yres);
  glutCreateWindow("mandelbrot");
  glewInit();
}

void pboinit() {
  glGenBuffers(1, &pbo);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
  glBufferData(GL_PIXEL_UNPACK_BUFFER, xres * yres * 4 * sizeof(GLubyte), 0, GL_STREAM_DRAW);
  glGenTextures(1, &tex);
  glBindTexture(GL_TEXTURE_2D, tex);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);
}

void exitfunc() {
  if (pbo) {
    hipGraphicsUnregisterResource(cuda_pbo_resource);
    glDeleteBuffers(1, &pbo);
    glDeleteTextures(1, &tex);
  }
}

void startkernel(uchar4* dout) {
  int xres = 1920, yres = 1080;
  int maxiter = 4096;
  float const cx0 = -2, cx1 = 1, cy0 = -1.2, cy1 = 1.2;
  float const cw = cx1 - cx0, ch = cy1 - cy0;
  float const recdiv = cw / float(xres), imcdiv = ch / float(yres);
  dim3 const dimBlock(32, 8);
  dim3 const dimGrid(std::ceil(float(xres) / dimBlock.x), std::ceil(float(yres) / dimBlock.y));
  calc<<<dimGrid, dimBlock>>>(dout, maxiter, recdiv, imcdiv, cx0, cy0, xres);
}

__global__ void calc(
    uchar4* dout, int maxiter, float recdiv, float imcdiv, float cx0, float cy0, int xres) {
  int iters = 0;
  float ztemp = 0, zreal = 0, zimag = 0;
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int ind = y * xres + x;
  float creal = x * recdiv + cx0;
  float cimag = y * imcdiv + cy0;
  while (iters <= maxiter && ((zimag * zimag) + (zreal * zreal)) <= 4.0) {
    // Update math algo, branch detection check
    ztemp = zreal;
    zreal = ((zreal * zreal) - (zimag * zimag));
    zimag = (ztemp * zimag);
    zimag += zimag;
    zreal += creal;
    zimag += cimag;
    iters += 1;
  }
  dout[ind].x = iters;
  dout[ind].y = iters;
  dout[ind].z = iters;
  dout[ind].w = iters;
}

void mandelbrot(int argc, char** argv) {
  int xres = 1920, yres = 1080;
  // GLUT STUFF
  initglut(&argc, argv);
  gluOrtho2D(0, xres, yres, 0);
  glutDisplayFunc(display);
  pboinit();
  glutMainLoop();
  atexit(exitfunc);
}

int main(int argc, char** argv) { mandelbrot(argc, argv); }
