#include "hip/hip_runtime.h"
// This is mostly a test
#include <GL/glew.h>
#include <GL/glut.h>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <immintrin.h>
#include <iostream>
#include <memory>
#include <stdio.h>
#include <thread>

int times = 0;
int xres = 1920, yres = 1080;
GLuint pbo = 0; // pixelbuffer obj
GLuint tex = 0; // texture obj
GLuint fbo = 0; // framebuffer obj
GLuint rbo = 0; // framebuffer obj
struct hipGraphicsResource* cuda_pbo_resource;
struct uchar4;

__global__ void calc(uchar4* rgba, int maxiter, float recdiv, float imcdiv, float cx0, float cy0,
    int xres, float* hist, int* iterimg, int& total);
__global__ void color(
    uchar4* rgba, float* hist, int xres, int* iterimg, float* huetest, int& total);
void pgm(int maxiter, int* img, int xres, int& yres);
void startkernel(uchar4* rgba);
void fboinit();
void pboinit();

void render() {
  uchar4* rgba = nullptr;
  hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
  hipGraphicsResourceGetMappedPointer((void**)&rgba, NULL, cuda_pbo_resource);
  startkernel(rgba);
  hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
}

void drawTex() {
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, xres, yres, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
  glEnable(GL_TEXTURE_2D);
  glBegin(GL_QUADS);
  glTexCoord2f(0.0f, 0.0f);
  glVertex2f(0, 0);
  glTexCoord2f(0.0f, 1.0f);
  glVertex2f(0, yres);
  glTexCoord2f(1.0f, 1.0f);
  glVertex2f(xres, yres);
  glTexCoord2f(1.0f, 0.0f);
  glVertex2f(xres, 0);
  glEnd();
  glDisable(GL_TEXTURE_2D);
}

void display() {
  render();
  drawTex();
  glutSwapBuffers();
}

void initglut(int* argc, char** argv) {
  glutInit(argc, argv);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
  glutInitWindowSize(xres, yres);
  glutCreateWindow("mandelbrot");
  glewInit();
  pboinit();
  //  fboinit();
}

void pboinit() {
  glGenBuffers(1, &pbo);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
  glBufferData(GL_PIXEL_UNPACK_BUFFER, xres * yres * 4 * sizeof(GLubyte), 0, GL_STREAM_DRAW);
  glGenTextures(1, &tex);
  glBindTexture(GL_TEXTURE_2D, tex);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);
}

void fboinit() {
  glGenFramebuffers(1, &fbo);
  glGenTextures(1, &tex);
  glGenRenderbuffers(1, &rbo);
  glBindFramebuffer(GL_FRAMEBUFFER, fbo);
  glBindTexture(GL_TEXTURE_2D, tex);
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, xres, yres, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
  glFramebufferTexture2D(GL_DRAW_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, tex, 0);
  glBindRenderbuffer(GL_RENDERBUFFER, rbo);
  glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH_COMPONENT24, xres, yres);
  glFramebufferRenderbuffer(GL_DRAW_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_RENDERBUFFER, rbo);
}

void exitfunc() {
  if (pbo) {
    hipGraphicsUnregisterResource(cuda_pbo_resource);
    glDeleteBuffers(1, &pbo);
    glDeleteTextures(1, &tex);
  }
}

void startkernel(uchar4* rgba) {
  int maxiter = 4096;
  float const cx0 = -2, cx1 = 1, cy0 = -1.2, cy1 = 1.2;
  float const cw = cx1 - cx0, ch = cy1 - cy0;
  float const recdiv = cw / float(xres), imcdiv = ch / float(yres);

  float* hist;
  int* img;
  hipMalloc(&img, xres * yres * sizeof(int));
  hipMalloc(&hist, xres * yres * sizeof(float));

  float* huetest;
  hipMalloc(&huetest, xres * yres * sizeof(float));

  int* total;
  hipMalloc(&total, sizeof(int));

  dim3 const dimBlock(32, 8);
  dim3 const dimGrid(std::ceil(float(xres) / dimBlock.x), std::ceil(float(yres) / dimBlock.y));
  calc<<<dimGrid, dimBlock>>>(rgba, maxiter, recdiv, imcdiv, cx0, cy0, xres, hist, img, *total);
  color<<<dimGrid, dimBlock>>>(rgba, hist, xres, img, huetest, *total);
  hipFree(img);
  hipFree(hist);
  hipFree(total);
  hipFree(huetest);
}

__global__ void calc(uchar4* rgba, int maxiter, float recdiv, float imcdiv, float cx0, float cy0,
    int xres, float* hist, int* img, int& total) {
  int iters = 0;
  float zmag2 = 0, zrsq = 0, zisq = 0, zreal = 0, zimag = 0;
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int ind = y * xres + x;
  float creal = x * recdiv + cx0;
  float cimag = y * imcdiv + cy0;

  while (iters <= maxiter && zmag2 <= 4) {
    zimag = ((zreal + zreal) * zimag) + cimag;
    zreal = (zrsq - zisq) + creal;
    zisq = zimag * zimag;
    zrsq = zreal * zreal;
    zmag2 = zrsq + zisq;
    iters++;
  }

  hist[iters]++;
  img[ind] = iters;
  // rgba[ind].x = 0; // R
  // rgba[ind].y = float(iters) * (255.0f / 4096.0f);
  // rgba[ind].z = 0;   // B
  // rgba[ind].w = 255; // A
}

__global__ void color(uchar4* rgba, float* hist, int xres, int* img, float* huetest, int& total) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int ind = y * xres + x;
  float hue;
  total = 0;

#if 1
  for (int i = 0; i < 4096; i++) {
    total += hist[i];
  }
#endif

  // TODO figure out why total changes
  float res = 96000;
  for (int i = 0; i <= img[ind]; i++) {
    hue += hist[i] / res;
  }

  rgba[ind].y = hue * 255;
  rgba[ind].x = 0;   // R
  rgba[ind].z = 0;   // B
  rgba[ind].w = 255; // A
}

void mandelbrot(int argc, char** argv) {
  // GLUT STUFF
  initglut(&argc, argv);
  gluOrtho2D(0, xres, yres, 0);
  glutDisplayFunc(display);
  glutMainLoop();
  atexit(exitfunc);
}

int main(int argc, char** argv) { mandelbrot(argc, argv); }
