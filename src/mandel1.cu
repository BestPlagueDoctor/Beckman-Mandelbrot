//This is mostly a test

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <immintrin.h>
#include <memory>
#include <new>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <thread>

__global__ void calc(int *img, int const maxiter, float const recdiv, float const imcdiv, float const cx0, float const cy0, int const xres);

void pgm(int maxiter, int* img, int xres, int yres);

void mandelbrot() {
  int* img;
  int xres = 3840, yres = 2160;
  int width = 3840, height = 2160, maxiter = 1000000;
  int* destimg = (int*)malloc(height * width * (sizeof(int)));
  size_t pitch;
  pitch = width * height * sizeof(int);
  hipMalloc(&img, width * sizeof(int) * height);
  float const cx0 = -2, cx1 = 1, cy0 = -1.2, cy1 = 1.2;
  float const cw = cx1 - cx0, ch = cy1 - cy0;
  float const recdiv = cw / float(xres), imcdiv = ch / float(yres);
  dim3 const dimBlock(16, 16);
  dim3 const dimGrid(std::ceil(float(xres) / dimBlock.x), std::ceil(float(yres) / dimBlock.y));
  auto const start = std::chrono::high_resolution_clock::now();
  calc<<<dimGrid, dimBlock>>>(img, maxiter, recdiv, imcdiv, cx0, cy0, xres);
  hipDeviceSynchronize();
  auto const end = std::chrono::high_resolution_clock::now();
  printf("Com time: %lums\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());
  hipMemcpy(destimg,img,pitch,hipMemcpyDeviceToHost);
  pgm(1000000, destimg, 3840, 2160);
  hipFree(img);
}

__global__ void calc(int *img, int const maxiter, float const recdiv, float const imcdiv, float const cx0, float const cy0, int const xres) {
  int iters = 0;
  float ztemp = 0, zreal = 0, zimag = 0;
  float creal = (blockIdx.x * blockDim.x + threadIdx.x) * recdiv + cx0;
  float cimag = (blockIdx.y * blockDim.y + threadIdx.y) * imcdiv + cy0;
  while (iters <= maxiter && ((zimag*zimag)+(zreal*zreal)) <= 4.0) {
    ztemp = zreal;
    zreal = ((zreal * zreal) - (zimag * zimag));
    zimag = (ztemp * zimag);
    zimag += zimag;
    zreal += creal;
    zimag += cimag;
    iters += 1;
  }
  img[(blockIdx.x * blockDim.x + threadIdx.x)+(blockIdx.y * blockDim.y + threadIdx.y)*xres] = iters;
}

//Needs editing to use cudaMemCopy
void pgm(int const maxiter, int* img, int const xres, int const yres) {
  const char filename[1024] = "smol.pgm";
  FILE* ofp;
  if ((ofp = fopen(filename, "w")) == NULL) {
        perror("FAILURE");
            return;

  }
  fprintf(ofp, "P2\n");
  fprintf(ofp, "%d %d \n", xres, yres);
  fprintf(ofp, "%d \n", maxiter);
  for (int i = 0; i < xres * yres; i++) {
    if (i % xres == 0) {
      fprintf(ofp, "\n");
    }
    fprintf(ofp, "%d ", img[i]);
  }
  free(img);
}


int main() {
  mandelbrot();
}
