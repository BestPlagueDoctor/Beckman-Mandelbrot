#include "hip/hip_runtime.h"
//This is mostly a test
#include <iostream>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <immintrin.h>
#include <memory>
#include <new>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <thread>
#include <GL/glew.h>
#define GLEW_STATIC
#include <SFML/System.hpp>
#include <SFML/Window.hpp>
#include <SFML/Graphics.hpp>
#include <cuda_gl_interop.h>

__global__ void calc(int *img, int const maxiter, float const recdiv, float const imcdiv, float const cx0, float const cy0, int const xres);

void pgm(int maxiter, int* img, int xres, int yres);

void mandelbrot() {
  int xres = 1920, yres = 1080;
  int width = 1920, height = 1080, maxiter = 4096;
  //GLEW STUFF
  sf::Window window(sf::VideoMode(1,1), "OpenGL");
  glewExperimental = GL_TRUE;
  glewInit();
  glViewport(0,0,xres,yres);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0,1.0f,0,1.0f,-1.0f,1.0f);
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();
  glEnable(GL_DEPTH_TEST);
  glClearColor(1.0f,1.0f,1.0f,1.5f);
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  cudaGLSetGLDevice(1);
  GLuint mainbuffer;
  glGenBuffers(1, &mainbuffer);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, mainbuffer);
  glBufferData(GL_PIXEL_UNPACK_BUFFER, xres * yres * sizeof(int), NULL, GL_DYNAMIC_COPY);
  cudaGLRegisterBufferObject(mainbuffer);
  glEnable(GL_TEXTURE_2D);
  unsigned int textureID;
  glGenTextures(1,&textureID);
  glBindTexture(GL_TEXTURE_2D,textureID);
  glTexImage2D(GL_TEXTURE_2D,0,GL_RGBA8,xres,yres,0,GL_BGRA,GL_UNSIGNED_BYTE,NULL);
  glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MIN_FILTER,GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MAG_FILTER,GL_LINEAR);

  //END GLEW STUFF
  int* img;
  hipGraphicsGLRegisterBuffer(img, mainbuffer);
  int* destimg = (int*)malloc(height * width * (sizeof(int)));
  size_t pitch;
  pitch = width * height * sizeof(int);
  hipMalloc(&img, width * sizeof(int) * height);
  //Coordinate update for more interesting images
  float const cx0 = -2, cx1 = 1, cy0 = -1.2, cy1 = 1.2;
  float const cw = cx1 - cx0, ch = cy1 - cy0;
  float const recdiv = cw / float(xres), imcdiv = ch / float(yres);
  dim3 const dimBlock(32, 8);
  dim3 const dimGrid(std::ceil(float(xres) / dimBlock.x), std::ceil(float(yres) / dimBlock.y));
  auto const start = std::chrono::high_resolution_clock::now();
  calc<<<dimGrid, dimBlock>>>(img, maxiter, recdiv, imcdiv, cx0, cy0, xres);
  hipDeviceSynchronize();
  auto const end = std::chrono::high_resolution_clock::now();
  printf("Com time: %lums\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());
  hipMemcpy(destimg,img,pitch,hipMemcpyDeviceToHost);
  pgm(maxiter, destimg, xres, yres);
  hipFree(img);
}

__global__ void calc(int *img, int const maxiter, float const recdiv, float const imcdiv, float const cx0, float const cy0, int const xres) {
  int iters = 0;
  float ztemp = 0, zreal = 0, zimag = 0;
  float creal = (blockIdx.x * blockDim.x + threadIdx.x) * recdiv + cx0;
  float cimag = (blockIdx.y * blockDim.y + threadIdx.y) * imcdiv + cy0;
  while (iters <= maxiter && ((zimag*zimag)+(zreal*zreal)) <= 4.0) {
    //Update math algo, branch detection check
    ztemp = zreal;
    zreal = ((zreal * zreal) - (zimag * zimag));
    zimag = (ztemp * zimag);
    zimag += zimag;
    zreal += creal;
    zimag += cimag;
    iters += 1;
  }
  img[(blockIdx.x * blockDim.x + threadIdx.x)+(blockIdx.y * blockDim.y + threadIdx.y)*xres] = iters;
}

//Needs editing to use cudaMemCopy
void pgm(int const maxiter, int* img, int const xres, int const yres) {
  const char filename[1024] = "smol.pgm";
  FILE* ofp;
  if ((ofp = fopen(filename, "w")) == NULL) {
        perror("FAILURE");
            return;

  }
  fprintf(ofp, "P2\n");
  fprintf(ofp, "%d %d \n", xres, yres);
  fprintf(ofp, "%d \n", maxiter);
  for (int i = 0; i < xres * yres; i++) {
    if (i % xres == 0) {
      fprintf(ofp, "\n");
    }
    fprintf(ofp, "%d ", img[i]);
  }
  free(img);
}


int main() {
  mandelbrot();
}
